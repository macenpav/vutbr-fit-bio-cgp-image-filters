#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <ctime>
#include <fstream>
#include <sstream>
#include <iomanip>

#include "cgp.cuh"
#include "cgp_cuda.cuh"

namespace imcgp
{ 
	float calc_fitness(FitnessMethod method, cv::Mat const& input, cv::Mat const& reference, const uint32 offset = 1)
	{
		if (input.rows != reference.rows || input.cols != reference.cols)
		{
			std::cerr << "Input image and reference image dimensions are different." << std::endl;
			return ERROR_FITNESS;
		}

		float fitness = ERROR_FITNESS;        
		switch (method)
		{
			case MDPP:
			{
				fitness = 0.f;
                for (uint32 y = offset; y < input.rows - offset; ++y)
				{
                    for (uint32 x = offset; x < input.cols - offset; ++x)
						fitness += std::abs(static_cast<float>(input.at<uint8>(y, x)) - static_cast<float>(reference.at<uint8>(y, x)));
				}
                fitness /= static_cast<float>((input.cols - 2 * offset) * (input.rows - 2 * offset));
				break;
			}
			case PSNR:
			{
				float tmp = 0.f;
                for (uint32 y = offset; y < input.rows - offset; ++y)
				{
                    for (uint32 x = offset; x < input.cols - offset; ++x)
					{
						float a = static_cast<float>(input.at<uint8>(y, x)) - static_cast<float>(reference.at<uint8>(y, x));
						tmp += (a * a);
					}
				}
                tmp /= ((input.cols - 2 * offset) * (input.rows - 2 * offset));
				fitness = 10.f * std::log10(255.f * 255.f / tmp);
				break;
			}
            case MSE:
            {
                fitness = 0.f;
                for (uint32 y = offset; y < input.rows - offset; ++y)
                {
                    for (uint32 x = offset; x < input.cols - offset; ++x)
                    {
                        float tmp = static_cast<float>(input.at<uint8>(y, x)) - static_cast<float>(reference.at<uint8>(y, x));
                        fitness += tmp * tmp;
                    }
                }
                fitness /= static_cast<float>((input.cols - 2 * offset) * (input.rows - 2 * offset));
                break;
            }
			
			default:			
				break;			
		}

		return fitness;
	}

	void get_3x3_kernel(uint8* kernel, cv::Mat const& input, uint32 const& x, uint32 const& y)
	{
		kernel[0] = input.at<uint8>(y - 1, x - 1);
		kernel[1] = input.at<uint8>(y - 1, x);
		kernel[2] = input.at<uint8>(y - 1, x + 1);

		kernel[3] = input.at<uint8>(y, x - 1);
		kernel[4] = input.at<uint8>(y, x);
		kernel[5] = input.at<uint8>(y, x + 1);

		kernel[6] = input.at<uint8>(y + 1, x - 1);
		kernel[7] = input.at<uint8>(y + 1, x);
		kernel[8] = input.at<uint8>(y + 1, x + 1);
	}

    void get_5x5_kernel(uint8* kernel, cv::Mat const& input, uint32 const& x, uint32 const& y)
    {
        kernel[0] = input.at<uint8>(y - 2, x - 2);
        kernel[1] = input.at<uint8>(y - 2, x - 1);
        kernel[2] = input.at<uint8>(y - 2, x);
        kernel[3] = input.at<uint8>(y - 2, x + 1);
        kernel[4] = input.at<uint8>(y - 2, x + 2);

        kernel[5] = input.at<uint8>(y - 1, x - 2);
        kernel[6] = input.at<uint8>(y - 1, x - 1);
        kernel[7] = input.at<uint8>(y - 1, x);
        kernel[8] = input.at<uint8>(y - 1, x + 1);
        kernel[9] = input.at<uint8>(y - 1, x + 2);

        kernel[10] = input.at<uint8>(y, x - 2);
        kernel[11] = input.at<uint8>(y, x - 1);
        kernel[12] = input.at<uint8>(y, x);
        kernel[13] = input.at<uint8>(y, x + 1);
        kernel[14] = input.at<uint8>(y, x + 2);

        kernel[15] = input.at<uint8>(y + 1, x - 2);
        kernel[16] = input.at<uint8>(y + 1, x - 1);
        kernel[17] = input.at<uint8>(y + 1, x);
        kernel[18] = input.at<uint8>(y + 1, x + 1);
        kernel[19] = input.at<uint8>(y + 1, x + 2);

        kernel[20] = input.at<uint8>(y + 2, x - 2);
        kernel[21] = input.at<uint8>(y + 2, x - 1);
        kernel[22] = input.at<uint8>(y + 2, x);
        kernel[23] = input.at<uint8>(y + 2, x + 1);
        kernel[24] = input.at<uint8>(y + 2, x + 2);
    }

    uint8 eval_chromosome(Chromosome const& chromosome, uint8* inputs, uint32 const& numRows, uint32 const& numCols, uint32 const& numInputs)
	{		
        uint8* outputs;
        if (numInputs == 9)
        { 
            outputs = (uint8*)malloc(CGP_PARAM_TOTAL_3X3);
            memcpy(outputs, inputs, CGP_PARAM_INPUTS_3X3 * sizeof(uint8));
        }        
        else if (numInputs == 25)
        {
            outputs = (uint8*)malloc(CGP_PARAM_TOTAL_5X5);
            memcpy(outputs, inputs, CGP_PARAM_INPUTS_5X5 * sizeof(uint8));
        }
		
		
		uint32 in1, in2, func;

		uint32 v = 0;

		uint8 out;
		for (uint32 i = 0; i < numCols; i++)
		{
			for (uint32 j = 0; j < numRows; j++)
			{
				in1 = outputs[chromosome.val[v++]];
				in2 = outputs[chromosome.val[v++]];
				func = chromosome.val[v++];

				switch (func)
				{
					case FUNC_CONST: out = 255; break;
					case FUNC_IDENTITY: out = in1; break;
					case FUNC_INVERT: out = 255 - in1; break;
					case FUNC_OR: out = in1 | in2; break;
					case FUNC_AND: out = in1 & in2; break;
					case FUNC_NAND: out = ~(in1 & in2); break;
					case FUNC_XOR: out = in1 ^ in2; break;
					case FUNC_SHR1: out = in1 >> 1; break;
					case FUNC_SHR2: out = in1 >> 2; break;
					case FUNC_SWAP: out = ((in1 & 0x0F) << 4) | (in2 & 0x0F); break;					
					case FUNC_ADD:
					{
						if (static_cast<uint32>(in1)+static_cast<uint32>(in2) > 255)
							out = 255;
						else
							out = in1 + in2;
						break;
					}
					case FUNC_AVERAGE:
					{
						out = static_cast<uint8>((static_cast<uint32>(in1)+static_cast<uint32>(in2)) >> 1);
						break;
					}
					case FUNC_MAX: out = std::max(in1, in2); break;
					case FUNC_MIN: out = std::min(in1, in2); break;
                    case FUNC_SHL1: out = in1 << 1; break;
                    case FUNC_SHL2: out = in1 << 2; break;
					default: out = 255;
				}

                if (numInputs == 9)                
                    outputs[numRows * i + j + CGP_PARAM_INPUTS_3X3] = out;                
                else if (numInputs == 25)
                    outputs[numRows * i + j + CGP_PARAM_INPUTS_5X5] = out;				
			}
		}

		return out;
	}

	void evolve_population(Population& population, std::vector<uint32>* possibleValues, uint32 const& bestFilter, uint32 const& numPopulation, uint32 const& numMutate, uint32 const& numRows, uint32 const& numCols, uint32 const& numInputs)
	{
		Chromosome parent = population[bestFilter];

		population[0] = parent;
		for (uint32 ch = 1; ch < numPopulation; ++ch)
		{
            population[ch] = mutate(parent, possibleValues, numMutate, CGP_CHROMOSOME_SIZE, numRows, numCols, numInputs);
		}
	}

	void find_possible_col_values(std::vector<uint32>* table, uint32 const& numRows, uint32 const& numCols, uint32 const& lback, uint32 const& numInputs)
	{
		for (uint32 i = 0; i < numCols; ++i)
		{
            uint32 minidx = numRows * (i - lback) + numInputs;
            if (minidx < numInputs)
                minidx = numInputs;

            uint32 maxidx = i * numRows + numInputs;

            for (uint32 j = 0; j < numInputs; ++j)
				table[i].push_back(j);

			for (uint32 j = minidx; j < maxidx; ++j)
				table[i].push_back(j);
		}

		#ifdef DEBUG
		std::cout << "Possible Values:" << std::endl;
		std::cout << "-------------------------------" << std::endl;
		for (uint32 i = 0; i < numCols; ++i)
		{
			std::cout << "Col " << i << " vals: ";
			std::vector<uint32> colVals = table[i];
			for (std::vector<uint32>::const_iterator it = colVals.begin(); it != colVals.end(); ++it)
			{
				std::cout << *it << " ";
			}
			std::cout << std::endl;
		}
		std::cout << "-------------------------------" << std::endl;
		#endif
	}

	void create_init_population(Population& population, std::vector<uint32>* possibleValues, uint32 const& maxPopulation, uint32 const& numRows, uint32 const& numCols, uint32 const& numInputs)
	{
		population.reserve(maxPopulation);
		for (uint32 i = 0; i < maxPopulation; ++i)
		{
			Chromosome ch;
			uint32 j = 0;
			for (uint32 col = 0; col < numCols; ++col)
			{
				for (uint32 row = 0; row < numRows; ++row)
				{
					ch.val[j++] = possibleValues[col][rand() % possibleValues[col].size()];
					ch.val[j++] = possibleValues[col][rand() % possibleValues[col].size()];
					ch.val[j++] = rand() % NUM_FUNCTIONS;
				}
			}

			for (uint32 output = 0; output < CGP_PARAM_OUTPUTS; ++output)
                ch.val[j++] = rand() % (numRows * numCols + numInputs);

			population[i] = ch;
		}

		#ifdef DEBUG
		std::cout << std::endl << "Initial population:" << std::endl;
		std::cout << "-------------------------------" << std::endl;
		for (uint32 i = 0; i < maxPopulation; ++i)
		{
			Chromosome ch = population[i];
			for (uint32 j = 0; j < CGP_CHROMOSOME_SIZE; ++j)
			{
				std::cout << ch.val[j];
				if (j % 3 < 2)
					std::cout << ",";
				else
					std::cout << ";";
			}			
			std::cout << std::endl;
		}
		std::cout << "-------------------------------" << std::endl;
		#endif
	}

	Chromosome mutate(Chromosome parent, const std::vector<uint32>* possibleValues, uint32 const& numBits, uint32 const& chromosomeLength, uint32 const& numRows, uint32 const& numCols, uint32 const& numInputs)
	{
		const uint32 numGenes = rand() % numBits + 1;
		for (uint32 i = 0; i < numGenes; ++i)
		{		
			const uint32 idx = rand() % ((3 * numRows * numCols) + CGP_PARAM_OUTPUTS);
			const uint32 col = idx / (3 * numRows);
			const uint32 rnd = rand();
			
			// ouptut
			if (idx < (3 * numRows * numCols))
			{		
				// input
				if ((idx % 3) < 2)
					parent.val[idx] = possibleValues[col][rnd % possibleValues[col].size()];
				// func
				else
					parent.val[idx] = rand() % NUM_FUNCTIONS;				
			}
			else			
                parent.val[idx] = rand() % (numCols * numRows + numInputs);
			
		}
		return parent;
	}

	///////////////////////////////////////////////////////////////
	// CGPWrapper
	///////////////////////////////////////////////////////////////

	bool CGPWrapper::run(FitnessMethod method, uint32 const& numRuns, uint32 const& numGenerations, uint32 const& numPopulation, uint32 const& numMutate, uint32 const& numInputs)
	{
		srand(time(NULL));

		if (_inputImage.rows != _referenceImage.rows || _inputImage.cols != _referenceImage.cols)
		{
			std::cerr << "Input image and reference image dimensions are different." << std::endl;
			return false;
		}      

        if (numInputs != 9 && numInputs != 25)
        {            
            std::cerr << "The number of inputs " << std::endl;
            return false;
        }

		_filteredImage = cv::Mat(_inputImage.rows, _inputImage.cols, CV_8UC1);
        
        if (_options & OPT_CUDA_ACCELERATION)
        {
            GPU_CHECK_ERROR(hipMalloc((void**)&_cudaInputImage, _inputImage.rows * _inputImage.cols * sizeof(uint8)));
            GPU_CHECK_ERROR(hipMalloc((void**)&_cudaFilteredImage, _inputImage.rows * _inputImage.cols * sizeof(uint8)));            
            GPU_CHECK_ERROR(hipMemcpy(_cudaInputImage, _inputImage.data, _inputImage.cols * _inputImage.rows * sizeof(uint8), hipMemcpyHostToDevice));
        }

		std::vector<uint32> possibleValues[CGP_PARAM_COLS];
        find_possible_col_values(possibleValues, CGP_PARAM_ROWS, CGP_PARAM_COLS, CGP_PARAM_LBACK, numInputs);

        // in case we are using gpu acceleration
        // decide which kernel to use
        void(*filter_image_func)(const uint8*, uint8*, const Chromosome*, const uint32, const uint32);
        if (_options & OPT_CUDA_ACCELERATION)
        {
            if (numInputs == 9)
                filter_image_func = cuda::filter_image_3x3;
            else if (numInputs == 25)
                filter_image_func = cuda::filter_image_5x5;
            else
            {
                std::cerr << "Wrong input parameter value." << std::endl;
                return false;
            }
        }        

		for (uint32 r = 0; r < numRuns; ++r)
		{
            std::time_t t = std::time(NULL);
            auto now = std::time(nullptr);
            auto tm = *std::localtime(&now);
            std::stringstream ss;
            ss << std::put_time(&tm, "%d-%m-%Y %H-%M-%S");
            std::string date = ss.str();

            std::string outStatsFilename = "stats (" + date + ").txt";
            std::string outCsvFilename = "fitness (" + date + ").csv";

            if (_options & OPT_OUTPUT_CSV)
            {
                std::ofstream file;
                file.open(outCsvFilename, std::ios::out);
                file << "generation;fitness" << std::endl;
                file.close();
            }

			_stats.total_time = 0.0;
	
			// generate initial population					
			auto timer_init_start = std::chrono::high_resolution_clock::now();
			Population pop;
			create_init_population(pop, possibleValues, numPopulation, CGP_PARAM_ROWS, CGP_PARAM_COLS, numInputs);
            auto timer_init_end = std::chrono::high_resolution_clock::now();

            _stats.init_time = static_cast<double>(std::chrono::duration_cast<std::chrono::milliseconds>(timer_init_end - timer_init_start).count()) / 1000.0;

			if (_options & OPT_VERBOSE)							
                std::cout << "Initial population created in " << _stats.init_time << " ms" << std::endl;             

			float fitness;
			switch (method)
			{
				// max value
                case MSE:
				case MDPP:
					fitness = std::numeric_limits<float>::max();
					break;
				// min value
				case PSNR:
					fitness = std::numeric_limits<float>::min();
					break;
			}

			for (uint32 gen = 0; gen < numGenerations; ++gen)
			{			
				auto timer_gen_start = std::chrono::high_resolution_clock::now();

				int32 bestFilter = 0;
				std::vector<uint32> candidates;		
                Chromosome* cuda_pop;                

                if (_options & OPT_CUDA_ACCELERATION)
                {
                    GPU_CHECK_ERROR(hipMalloc((void**)&cuda_pop, sizeof(Chromosome) * numPopulation));
                    GPU_CHECK_ERROR(hipMemcpy(cuda_pop, &pop[0], sizeof(Chromosome) * numPopulation, hipMemcpyHostToDevice));
                }

				for (uint32 ch = 0; ch < numPopulation; ++ch)
				{					
					// generate filtered image using chromosome evaluation
				
                    if (_options & OPT_CUDA_ACCELERATION)
                    {
                        dim3 block(16, 16);
                        dim3 grid(64, 64);                        
                        
                        filter_image_func<<<grid, block>>>(_cudaInputImage, _cudaFilteredImage, &cuda_pop[ch], _inputImage.cols, _inputImage.rows);                      

                        GPU_CHECK_ERROR(hipMemcpy(_filteredImage.data, _cudaFilteredImage, _filteredImage.cols * _filteredImage.rows * sizeof(uint8), hipMemcpyDeviceToHost));
                    }
                    else
                    {
                        for (uint32 y = 1; y < _inputImage.rows - 1; ++y)
                        {
                            for (uint32 x = 1; x < _inputImage.cols - 1; ++x)
                            {
                                // get image kernel and copy it to outputs
                                if (numInputs == CGP_PARAM_INPUTS_3X3)
                                {
                                    uint8 kernel[CGP_PARAM_INPUTS_3X3];
                                    get_3x3_kernel(kernel, _inputImage, x, y);
                                    _filteredImage.at<uint8>(y, x) = eval_chromosome(pop[ch], kernel, CGP_PARAM_ROWS, CGP_PARAM_COLS, numInputs);
                                }
                                else if (numInputs == CGP_PARAM_INPUTS_5X5)
                                {
                                    uint8 kernel[CGP_PARAM_INPUTS_5X5];
                                    get_3x3_kernel(kernel, _inputImage, x, y);
                                    _filteredImage.at<uint8>(y, x) = eval_chromosome(pop[ch], kernel, CGP_PARAM_ROWS, CGP_PARAM_COLS, numInputs);
                                }   
                                else
                                {
                                    std::cerr << "Wrong input parameter value." << std::endl;
                                    return false;
                                }
                            }
                        }
                    }

                    float newFitness = calc_fitness(method, _filteredImage, _referenceImage, numInputs == 9 ? 1 : 2);                    
					
					if (newFitness == ERROR_FITNESS)
					{
						std::cerr << "An error occured while calculating fitness." << std::endl;
						return false;
					}
                

					switch (method)
					{
						// min value
                        case MSE:
						case MDPP: // mean difference per pixel
						{
							if (newFitness < fitness)
							{
								candidates.clear();
								candidates.push_back(ch);
								fitness = newFitness;
							}
							else if (newFitness == fitness)						
								candidates.push_back(ch);
														
							break;
						}
						// max value
						case PSNR: // peak signal-to-noise ration
						{
							if (newFitness > fitness)
							{
								candidates.clear();
								candidates.push_back(ch);
								fitness = newFitness;
							}
							else if (newFitness == fitness)
								candidates.push_back(ch);
							
							break;
						}                
					}										                    
				}

                if (_options & OPT_CUDA_ACCELERATION)
                {
                    hipFree(cuda_pop);
                }                

				bestFilter = candidates[rand() % candidates.size()];
				evolve_population(pop, possibleValues, bestFilter, numPopulation, numMutate, CGP_PARAM_ROWS, CGP_PARAM_COLS, numInputs);			
				                                
                auto timer_gen_end = std::chrono::high_resolution_clock::now();
                double gen_time = static_cast<double>(std::chrono::duration_cast<std::chrono::milliseconds>(timer_gen_end - timer_gen_start).count()) / 1000.0;

                _stats.total_time += gen_time;
                _stats.average_gen_time += gen_time;
                
				if (_options & OPT_VERBOSE)
				{					       
                    std::cout << "Generation (" << gen << ") evaulated in " << gen_time << " ms" << std::endl;
                    std::cout << "Current fitness: " << fitness << std::endl;
				}		

                if (_options & OPT_OUTPUT_CSV)
                {
                    if (gen % 10 == 0)
                    {
                        std::ofstream file;
                        file.open(outCsvFilename, std::ios::out | std::ios::app);
                        file << gen << ";" << fitness << std::endl;
                        file.close();
                    }
                }
			}
														                
			_stats.average_gen_time /= static_cast<double>(numGenerations);
			_stats.fitness = fitness;
			_stats.best_filter = pop[0];
			_stats.num_generations = numGenerations;
			_stats.num_genes_mutated = numMutate;
            _stats.num_inputs = numInputs;
			_stats.population_size = numPopulation;
			_stats.method = method;			
            _stats.input_file = _inputImageFilename;
            _stats.reference_file = _refImageFilename;

			write_stats(outStatsFilename);
			save_image("filtered (" + date + ").jpg", FILTERED_IMAGE);
			save_image("original (" + date + ").jpg", ORIGINAL_IMAGE);
			save_image("reference (" + date + ").jpg", REFERENCE_IMAGE);
						
		}

        if (_options & OPT_CUDA_ACCELERATION)
        {
            GPU_CHECK_ERROR(hipFree(_cudaInputImage));
            GPU_CHECK_ERROR(hipFree(_cudaFilteredImage));            
        }

		return true;
	}

	void CGPWrapper::write_stats(std::string const& filename)
	{
		std::ofstream myfile;
		myfile.open(filename);

        myfile << "Input image: " << _stats.input_file << std::endl;
        myfile << "Reference image: " << _stats.reference_file << std::endl;
        myfile << "Init time: " << _stats.init_time << std::endl;
		myfile << "Total time: " << _stats.total_time << std::endl;
		myfile << "Average gen. time: " << _stats.average_gen_time << std::endl;
		myfile << "Fitness method: ";
		switch (_stats.method)
		{
            case MSE:
                myfile << "MSE";
                break;        
			case PSNR:
				myfile << "PSNR";
				break;
			case MDPP:
				myfile << "MDPP";
				break;
		}
		myfile << std::endl;
		myfile << "Fitness: " << _stats.fitness << std::endl;
		myfile << "Best filter: ";

		for (uint32 i = 0; i < CGP_CHROMOSOME_SIZE; ++i)
		{
			myfile << _stats.best_filter.val[i];
			if (i != CGP_CHROMOSOME_SIZE - 1)
			{
				if (i % 3 < 2)
					myfile << ",";
				else
					myfile << ";";
			}			
		}
		myfile << std::endl;
        myfile << "Number of inputs: " << _stats.num_inputs << std::endl;
		myfile << "Number of generations: " << _stats.num_generations << std::endl;
		myfile << "Max. genes mutated: " << _stats.num_genes_mutated << std::endl;
		myfile << "Population size: " << _stats.population_size << std::endl;		

		myfile.close();
	}

	bool CGPWrapper::load_image(std::string const& filename, ImageType type)
	{
		switch (type)
		{
			case REFERENCE_IMAGE:
			{
				_referenceImage = cv::imread(filename, CV_LOAD_IMAGE_GRAYSCALE);
				if (!_inputImage.data)
					return false;
                _refImageFilename = filename;
				break;
			}
			case ORIGINAL_IMAGE:
			{
				_inputImage = cv::imread(filename, CV_LOAD_IMAGE_GRAYSCALE);
				if (!_inputImage.data)
					return false;
                _inputImageFilename = filename;
				break;
			}
			default:
				break;
		}		        

		return true;
	}

	void CGPWrapper::display_image(ImageType type)
	{
		switch (type)
		{
			case REFERENCE_IMAGE:
				cv::imshow("Reference image", _referenceImage);			
				break;
			case FILTERED_IMAGE:
				cv::imshow("Filtered image", _filteredImage);
				break;
			case ORIGINAL_IMAGE:
				cv::imshow("Original image", _inputImage);
				break;
			default:
				break;
		}		
		cv::waitKey(0);
	}
	
	void CGPWrapper::save_image(std::string const& filename, ImageType type)
	{
		switch (type)
		{
			case REFERENCE_IMAGE:
				cv::imwrite(filename, _referenceImage);
				break;
			case FILTERED_IMAGE:
				cv::imwrite(filename, _filteredImage);
				break;
			case ORIGINAL_IMAGE:
				cv::imwrite(filename, _inputImage);
				break;
			default:
				break;
		}		
	}

	void CGPWrapper::set_options(uint32 const& opts)
	{
		_options = opts;
	}

} // namespace CGP
